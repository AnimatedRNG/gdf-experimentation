#include "hip/hip_runtime.h"
#include "hip/hip_vector_types.h"
#include "cuda_matmul.hpp"

template <typename T>
__global__
void adam(size_t num_elements,
          T* params,
          const T* gradient,
          T* exp_avg,
          T* exp_avg_sq,
          const float lr,
          const float beta_1,
          const float beta_2,
          const float weight_decay,
          const float eps,
          const int iteration) {
          
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (index >= num_elements) {
        return;
    }
    
    float grad = gradient[index] + weight_decay * params[index];
    exp_avg[index] = exp_avg[index] * beta_1 + (1.0f - beta_1) * grad;
    exp_avg_sq[index] = exp_avg_sq[index] * beta_2 + (1.0f - beta_2) * grad * grad;
    
    float denom = sqrtf(exp_avg_sq[index]) + eps;
    
    float bias_correction_1 = 1.0f / (1.0f - powf(beta_1, (float) iteration));
    float bias_correction_2 = 1.0f / (1.0f - powf(beta_2, (float) iteration));
    
    float adapted_learning_rate = lr * bias_correction_1 / sqrtf(bias_correction_2);
    params[index] = params[index] - adapted_learning_rate * exp_avg[index] / denom;
}

template <typename T, size_t N>
class AdamOptimizer {
  public:
  
    explicit AdamOptimizer(T* params_device,
                           T* gradient_device,
                           cuda_array<T, N>* params_host,
                           cuda_array<T, N>* gradient_host,
                           T zero_value,
                           const float& lr = 1e-3f,
                           const float& beta_1 = 0.9f,
                           const float& beta_2 = 0.99f,
                           const float& weight_decay = 0.0f,
                           const float& eps = 1e-8f) :
        params_device_(params_device),
        gradient_device_(gradient_device),
        params_host_(params_host),
        gradient_host_(gradient_host),
        lr_(lr), beta_1_(beta_1), beta_2_(beta_2),
        weight_decay_(weight_decay), eps_(eps),
        iterations(0),
        num_elements(params_host->num_elements) {
        
        for (int i = 0; i < N; i++) {
            assert(params_host->shape[i] == gradient_host->shape[i]);
        }
        
        std::vector<T> zeros(params_host->num_elements, zero_value);
        
        size_t buf_size = params_host->num_elements * sizeof(T);
        hipMalloc(&exp_avg_device_, buf_size);
        hipMalloc(&exp_avg_sq_device_, buf_size);
        hipMemcpy(exp_avg_device_, &(zeros[0]), buf_size, hipMemcpyHostToDevice);
        hipMemcpy(exp_avg_sq_device_, &(zeros[0]), buf_size, hipMemcpyHostToDevice);
    }
    
    ~AdamOptimizer() {
        hipFree(exp_avg_device_);
        hipFree(exp_avg_sq_device_);
    }
    
    void step() {
        size_t block_size = 64;
        
        size_t grid_size = (size_t) ceil((float) num_elements /
                                         (float) block_size);
                                         
        adam<T> <<< grid_size, block_size >>> (num_elements,
                                               params_device_,
                                               gradient_device_,
                                               exp_avg_device_,
                                               exp_avg_sq_device_,
                                               
                                               lr_,
                                               beta_1_,
                                               beta_2_,
                                               weight_decay_,
                                               eps_,
                                               ++iterations);
    }
    
  private:
    float lr_;
    float beta_1_;
    float beta_2_;
    float weight_decay_;
    float eps_;
    int iterations;
    
    size_t num_elements;
    
    T* params_device_;
    T* gradient_device_;
    
    cuda_array<T, N>* params_host_;
    cuda_array<T, N>* gradient_host_;
    
    T* exp_avg_device_;
    T* exp_avg_sq_device_;
};

int main() {
    size_t dims[2] = {5, 2};
    size_t* dims_device;
    
    float expected_params_m[2][5] = {
        {0.9990, 1.0010, 0.9990, 0.9990, 1.0010},
        {1.0010, 0.9990, 0.9990, 0.9990, 1.0010}
    };
    float initial_params_m[2][5] = {
        {1.0f, 1.0f, 1.0f, 1.0f, 1.0f},
        {1.0f, 1.0f, 1.0f, 1.0f, 1.0f}
    };
    
    float initial_grad_m[2][5] = {
        {0.02574085, -0.26188788, 0.5158403, 0.5158403, -10.2624},
        {-0.26188788, 0.02574085, 0.5158403, 0.5158403, -10.2624}
    };
    
    cuda_array<float, 2> initial_params;
    assign(&initial_params, (float*)initial_params_m, dims);
    
    cuda_array<float, 2> initial_grad;
    assign(&initial_grad, (float*)initial_grad_m, dims);
    
    float* initial_params_device = to_device<float, 2>(&initial_params,
                                   &dims_device);
    float* initial_grad_device = to_device<float, 2>(&initial_grad, &dims_device);
    
    AdamOptimizer<float, 2> adam(initial_params_device, initial_grad_device,
                                 &initial_params, &initial_grad,
                                 0.0f);
    adam.step();
    adam.step();
    
    to_host(initial_params_device, &initial_params);
    
    for (int i = 0; i < 2; i++) {
        for (int j = 0; j < 5; j++) {
            std::cout << index(&initial_params, j, i) << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}